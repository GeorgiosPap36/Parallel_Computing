#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

void Get_args(int argc, char* argv[], int* threads, int* blocks, int* r, int* c)  {
    *r = strtol(argv[1], NULL, 10);
    *c = strtol(argv[2], NULL, 10);
    *threads = strtol(argv[3], NULL, 10);
    *blocks = strtol(argv[4], NULL, 10);
}

double rand_double() 
{
    double random_value = (double)rand() / RAND_MAX;
    return random_value;
}

void Gen_matrix(double* A, int rows, int cols) {
    for (int i = 0; i < rows; i++)
        for (int j = 0; j < cols; j++)
                A[i*cols+j] = rand_double();
}

void Print_matrix(double* A, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++)
            printf("%4.3f ", A[i*cols + j]);
        printf("\n");
    }
}

void Correct_multiplication(int rows, int cols, double* A, double* B, double* C){
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < rows; j++) {
            C[i*rows + j] = 0;

            for (int k = 0; k < cols; k++) {
                C[i*rows + j] += A[i*cols + k] * B[k*rows + j];
            }
        }
    }
}

int Check_correctness(double* C, double* C_correct, int rows){
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < rows; j++){
            if (abs(C[i*rows + j] - C_correct[i*rows + j]) > 0.0000000001){
                printf("\n(%d, %d), (%f, %f)\n", i, j, C[i*rows + j], C_correct[i*rows + j]);
                return 0;
            }
        }
    }
    return 1;
}

__global__ void Mult_mats(int rows, int cols, double* device_A, double* device_B, double* device_C){
    int my_el = blockDim.x * blockIdx.x + threadIdx.x;

    if (my_el < rows * rows){
        int r = my_el / rows;
        int c = my_el % rows;
        device_C[my_el] = 0;
        for (int i = 0; i < cols; i++){
            device_C[my_el] += device_A[r * cols + i] * device_B[i * rows + c];
        }
    }   
}

int main(int argc, char* argv[]) {
    int threads_per_block, block_count, rows, cols;

    double* A; //rows x cols matrix
    double* B; //cols x rows matrix
    double* C; //A x B = C, rows x rows matrix
    //double* C_correct; // A x B = C_correct, the correct result of the multiplication
    
    double* device_A;
    double* device_B;
    double* device_C;

    clock_t start, finish;

    Get_args(argc, argv, &threads_per_block, &block_count, &rows, &cols);
    
    //Allocate memory on CPU for the matrices
    A = (double*) malloc(rows*cols*sizeof(double));
    B = (double*) malloc(cols*rows*sizeof(double));
    C = (double*) malloc(rows*rows*sizeof(double));
    //C_correct = (double*) malloc(rows*rows*sizeof(double));

    //Set the seed, for the generation of the random matrices
    srand(time(NULL)); 

    //Generate A matrix
    Gen_matrix(A, rows, cols);
    //printf("\nMatrix A(%d x %d): \n", rows, cols);
    //Print_matrix(A, rows, cols);

    //Generate B matrix
    Gen_matrix(B, cols, rows);
    //printf("\nMatrix B(%d x %d): \n", cols, rows);
    //Print_matrix(B, cols, rows);

    //Allocate memory on the GPU for the matrices
    hipMalloc((void**)&device_A, rows*cols*sizeof(double));
    hipMalloc((void**)&device_B, cols*rows*sizeof(double));
    hipMalloc((void**)&device_C, rows*rows*sizeof(double));

    //Copy the values of matrices A and B from the CPU to the GPU
    hipMemcpy(device_A, A, rows*cols*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(device_B, B, cols*rows*sizeof(double), hipMemcpyHostToDevice);

    start = clock(); //start timer

    Mult_mats <<<block_count, threads_per_block>>>(rows, cols, device_A, device_B, device_C); 
    hipDeviceSynchronize();

    finish = clock(); //end timer

    //Copy the values of C from the GPU to the CPU
    hipMemcpy(C, device_C, rows*rows*sizeof(double), hipMemcpyDeviceToHost);

    printf("\n----------------------\n");
    //printf("\nMatrix C(%d x %d): \n", rows, rows);
    //Print_matrix(C, rows, rows);

    //Correct_multiplication(rows, cols, A, B, C_correct);
    //printf("\nMatrix C_correct(%d x %d): \n", rows, rows);
    //Print_matrix(C_correct, rows, rows);

    //printf("\nCUDA solution is: %d (0 is incorrect, 1 is correct)\n", Check_correctness(C, C_correct, rows));

    printf("\n----------------------\n");
    printf("Runtime = %f\n", (double)(finish - start) / CLOCKS_PER_SEC);

    //Free memory
    free(A);
    free(B);
    free(C);
    //free(C_correct);
    hipFree(device_A);
    hipFree(device_B);
    hipFree(device_C);

    return 0;
}